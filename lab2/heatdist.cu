/*
 * This file contains the code for doing the heat distribution problem.
 * You do not need to modify anything except starting  gpu_heat_dist() at the bottom
 * of this file.
 * In gpu_heat_dist() you can organize your data structure and the call to your
 * kernel(s), memory allocation, data movement, etc.
 *
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

/* To index element (i,j) of a 2D square array of dimension NxN stored as 1D
   index(i, j, N) means access element at row i, column j, and N is the dimension which is NxN */
#define index(i, j, N) ((i) * (N)) + (j)

/*****************************************************************/

// Function declarations: Feel free to add any functions you want.
void seq_heat_dist(float *, unsigned int, unsigned int);
void gpu_heat_dist(float *, unsigned int, unsigned int);

/*****************************************************************/
/**** Do NOT CHANGE ANYTHING in main() function ******/

int main(int argc, char *argv[])
{
    unsigned int N;         /* Dimention of NxN matrix */
    int type_of_device = 0; // CPU or GPU
    int iterations = 0;
    int i;

    /* The 2D array of points will be treated as 1D array of NxN elements */
    float *playground;

    // to measure time taken by a specific part of the code
    double time_taken;
    clock_t start, end;

    if (argc != 4)
    {
        fprintf(stderr, "usage: heatdist num  iterations  who\n");
        fprintf(stderr, "num = dimension of the square matrix (50 and up)\n");
        fprintf(stderr, "iterations = number of iterations till stopping (1 and up)\n");
        fprintf(stderr, "who = 0: sequential code on CPU, 1: GPU version\n");
        exit(1);
    }

    type_of_device = atoi(argv[3]);
    N = (unsigned int)atoi(argv[1]);
    iterations = (unsigned int)atoi(argv[2]);

    /* Dynamically allocate NxN array of floats */
    playground = (float *)calloc(N * N, sizeof(float));
    if (!playground)
    {
        fprintf(stderr, " Cannot allocate the %u x %u array\n", N, N);
        exit(1);
    }

    /* Initialize it: calloc already initalized everything to 0 */
    // Edge elements  initialization
    for (i = 0; i < N; i++)
        playground[index(0, i, N)] = 100;
    for (i = 0; i < N; i++)
        playground[index(N - 1, i, N)] = 150;
    for (i = 1; i < N - 1; i++)
        playground[index(i, 0, N)] = 80;
    for (i = 1; i < N - 1; i++)
        playground[index(i, N - 1, N)] = 80;

    switch (type_of_device)
    {
    case 0:
        printf("CPU sequential version:\n");
        start = clock();
        seq_heat_dist(playground, N, iterations);
        end = clock();
        break;

    case 1:
        printf("GPU version:\n");
        start = clock();
        gpu_heat_dist(playground, N, iterations);
        hipDeviceSynchronize();
        end = clock();
        break;

    default:
        printf("Invalid device type\n");
        exit(1);
    }

    time_taken = ((double)(end - start)) / CLOCKS_PER_SEC;

    printf("Time taken = %lf\n", time_taken);

    free(playground);

    return 0;
}

/*****************  The CPU sequential version (DO NOT CHANGE THAT) **************/
void seq_heat_dist(float *playground, unsigned int N, unsigned int iterations)
{
    // Loop indices
    int i, j, k;
    int upper = N - 1;

    // number of bytes to be copied between array temp and array playground
    unsigned int num_bytes = 0;

    float *temp;
    /* Dynamically allocate another array for temp values */
    /* Dynamically allocate NxN array of floats */
    temp = (float *)calloc(N * N, sizeof(float));
    if (!temp)
    {
        fprintf(stderr, " Cannot allocate temp %u x %u array\n", N, N);
        exit(1);
    }

    num_bytes = N * N * sizeof(float);

    /* Copy initial array in temp */
    memcpy((void *)temp, (void *)playground, num_bytes);

    for (k = 0; k < iterations; k++)
    {
        /* Calculate new values and store them in temp */
        for (i = 1; i < upper; i++)
            for (j = 1; j < upper; j++)
                temp[index(i, j, N)] = (playground[index(i - 1, j, N)] +
                                        playground[index(i + 1, j, N)] +
                                        playground[index(i, j - 1, N)] +
                                        playground[index(i, j + 1, N)]) /
                                       4.0;

        /* Move new values into old values */
        memcpy((void *)playground, (void *)temp, num_bytes);
    }
}

__global__ void heatDistKernel(float *playground, float *temp, unsigned int N)
{
    int threadX = blockIdx.x * blockDim.x + threadIdx.x;
    int threadY = blockIdx.y * blockDim.y + threadIdx.y;
    int strideX = blockDim.x * gridDim.x;
    int strideY = blockDim.y * gridDim.y;

    for (int i = threadX; i < N; i += strideX)
    {
        for (int j = threadY; j < N; j += strideY)
        {
            if (i > 0 && i < N - 1 && j > 0 && j < N - 1)
            {
                temp[index(i, j, N)] = (playground[index(i - 1, j, N)] +
                                        playground[index(i + 1, j, N)] +
                                        playground[index(i, j - 1, N)] +
                                        playground[index(i, j + 1, N)]) /
                                       4.0;
            }
        }
    }
}

/***************** The GPU version: Write your code here *********************/
/* This function can call one or more kernels if you want ********************/
void gpu_heat_dist(float *playground, unsigned int N, unsigned int iterations)
{
#ifdef DEBUG
    printDeviceProperties();
#endif
    float *playground_device, *temp_device;
    size_t size = N * N * sizeof(float);

    hipMalloc(&playground_device, size);
    hipMalloc(&temp_device, size);

    hipMemcpy(playground_device, playground, size, hipMemcpyHostToDevice);
    hipMemcpy(temp_device, playground, size, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid(625, 625);

    for (int k = 0; k < iterations; ++k)
    {
        heatDistKernel<<<dimGrid, dimBlock>>>(playground_device, temp_device, N);
        hipDeviceSynchronize();

        float *temp = playground_device;
        playground_device = temp_device;
        temp_device = temp;
    }

    hipMemcpy(playground, playground_device, size, hipMemcpyDeviceToHost);

    hipFree(playground_device);
    hipFree(temp_device);
}

void printDeviceProperties()
{
    int device;
    hipDeviceProp_t properties;

    hipGetDevice(&device);
    hipGetDeviceProperties(&properties, device);

    printf("Max Threads per Block: %d\n", properties.maxThreadsPerBlock);
    printf("Max Block Dimensions: %d x %d x %d\n",
           properties.maxThreadsDim[0],
           properties.maxThreadsDim[1],
           properties.maxThreadsDim[2]);
    printf("Max Grid Dimensions: %d x %d x %d\n",
           properties.maxGridSize[0],
           properties.maxGridSize[1],
           properties.maxGridSize[2]);

    printf("Number of Streaming Multiprocessors (SMs): %d\n", properties.multiProcessorCount);
}
