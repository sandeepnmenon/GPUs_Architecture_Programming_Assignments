#ifdef USE_CHRONO
#include <chrono>
#else
#include <time.h>
#endif
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#define assertm(exp, msg) assert(((void)msg, exp))

int num_blocks, num_threads;

__global__ void matrixVectorMulKernel(int *M_d, int *V_d, int *R_d, size_t m);
void matrixVectorMul_cpu(int *M, int *V, int *R, size_t m);
void matrixVectorMul_gpu(int *M, int *V, int *R, size_t m);

void matrixVectorMul_cpu(int *M, int *V, int *R, size_t m)
{
    for (size_t i = 0; i < m; i++)
    {
        R[i] = 0;
        int sum_row = 0;
        for (size_t j = 0; j < m; j++)
        {
            sum_row += M[i * m + j] * V[j];
        }
        R[i] = sum_row;
    }
}

__global__ void matrixVectorMulKernel(int *M_d, int *V_d, int *R_d, size_t m)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (size_t i = threadId; i < m; i += stride)
    {
        int sum_row = 0;
        for (int j = 0; j < m; j++)
        {
            sum_row += M_d[i * m + j] * V_d[j];
        }
        R_d[i] = sum_row;
    }
}

void matrixVectorMul_gpu(int *M, int *V, int *R, size_t m)
{
    matrixVectorMulKernel<<<num_blocks, num_threads>>>(M, V, R, m);
    hipDeviceSynchronize();
}

void fill_random(int *M, int *V, size_t m)
{
    for (size_t i = 0; i < m; i++)
    {
        V[i] = rand() % INT8_MAX;
        for (int j = 0; j < m; j++)
        {
            M[i * m + j] = rand() % INT8_MAX;
        }
    }
}

double measure_time_func_cpu(void (*func)(int *, int *, int *, size_t), int *M, int *V, int *R, size_t m)
{
#ifdef USE_CHRONO
    std::chrono::time_point<std::chrono::high_resolution_clock> start = std::chrono::high_resolution_clock::now();
#else
    clock_t start = clock();
#endif

    func(M, V, R, m);

#ifdef USE_CHRONO
    std::chrono::time_point<std::chrono::high_resolution_clock> end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    return elapsed.count();
#else
    clock_t end = clock();
    return ((double)(end - start)) / CLOCKS_PER_SEC;
#endif
}

double measure_time_func_gpu(void (*func_gpu)(int *, int *, int *, size_t), int *M, int *V, int *R, int *M_device, int *V_device, int *R_device, size_t size_M, size_t size_V, size_t m)
{
    // Start timing
#ifdef USE_CHRONO
    std::chrono::time_point<std::chrono::high_resolution_clock> start = std::chrono::high_resolution_clock::now();
#else
    clock_t start = clock();
#endif
    hipError_t err;

    // Copy data to device
    err = hipMemcpy(M_device, M, size_M * sizeof(int), hipMemcpyHostToDevice);
    assertm(err == hipSuccess, "Failed to copy matrix M from host to device");

    err = hipMemcpy(V_device, V, size_V * sizeof(int), hipMemcpyHostToDevice);
    assertm(err == hipSuccess, "Failed to copy vector V from host to device");

    // Execute the kernel function
    func_gpu(M_device, V_device, R_device, m);

    // Copy result back to host
    err = hipMemcpy(R, R_device, size_V * sizeof(int), hipMemcpyDeviceToHost);
    assertm(err == hipSuccess, "Failed to copy vector R from device to host");

    // End timing
#ifdef USE_CHRONO
    std::chrono::time_point<std::chrono::high_resolution_clock> end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    return elapsed.count();
#else
    clock_t end = clock();
    return ((double)(end - start)) / CLOCKS_PER_SEC;
#endif
}

bool check_condition(bool cond, const char *msg)
{
    if (!cond)
        fprintf(stderr, "Error: %s\n", msg);

    return cond;
}

class IntArray
{
public:
    IntArray(size_t size) : data(NULL), size(size)
    {
        data = (int *)malloc(size * sizeof(int));
        if (!data)
            fprintf(stderr, "Error: Failed to allocate memory\n");
    }
    ~IntArray()
    {
        if (data)
            free(data);
    }

    int *data;
    size_t size;

    operator int *() { return data; }

    operator bool() { return data != NULL && size > 0; }

    int &operator[](size_t index) { return data[index]; }
    int operator[](size_t index) const { return data[index]; }

    bool operator==(const IntArray &other) const
    {
        if (size != other.size)
            return false;
        for (size_t i = 0; i < size; i++)
        {
            if (data[i] != other.data[i])
            {
                // printf("Mismatch at index %lu: %d != %d\n", i, data[i], other.data[i]);
                return false;
            }
        }

        return true;
    }

    bool operator!=(const IntArray &other) const { return !(*this == other); }
};

class IntArrayDevice
{
public:
    IntArrayDevice(size_t size) : data(NULL), size(size)
    {
        hipError_t err = hipMalloc(&data, size * sizeof(int));
        if (err != hipSuccess)
        {
            fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
            data = NULL;
        }
    }
    ~IntArrayDevice()
    {
        if (data)
            hipFree(data);
    }

    int *data;
    size_t size;

    operator int *() { return data; }

    operator bool() { return data != NULL && size > 0; }
};

void printArray(IntArray &arr)
{
    for (size_t i = 0; i < arr.size; i++)
    {
        printf("%d ", arr[i]);
    }
    printf("\n");
}

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        printf("Usage: %s <num_blocks> <num_threads>\n", argv[0]);
        return 1;
    }
    else
    {
        num_blocks = atoi(argv[1]);
        num_threads = atoi(argv[2]);
    }
    size_t test_dimensions[10] = {256, 512, 1024, 4096, 8192, 16384, 32768, 65536, 131072, 262144};
    // size_t test_dimensions[1] = {5};

    for (size_t dim_i = 0; dim_i < 10; ++dim_i)
    {
        size_t dimension = test_dimensions[dim_i];
        printf("Dimension: %lu\n", dimension);

        size_t size_M = dimension * dimension;
        IntArray M(size_M);
        if (!check_condition(M, "Memory allocation failed for M"))
            continue;

        size_t size_V = dimension;
        IntArray V(size_V);
        if (!check_condition(V, "Memory allocation failed for V"))
            continue;

        srand(time(0));
        fill_random(M, V, dimension);

        IntArray R(size_V);
        if (!check_condition(R, "Memory allocation failed for R"))
            continue;

        IntArrayDevice M_device(size_M);
        if (!check_condition(M_device, "Memory allocation failed for M_device"))
            continue;

        IntArrayDevice V_device(size_V);
        if (!check_condition(V_device, "Memory allocation failed for V_device"))
            continue;

        IntArrayDevice R_device(size_V);
        if (!check_condition(R_device, "Memory allocation failed for R_device"))
            continue;

        IntArray R_gpu(size_V);
        if (!check_condition(R_gpu, "Memory allocation failed for R_gpu"))
            continue;

        double cpu_time = measure_time_func_cpu(matrixVectorMul_cpu, M, V, R, dimension);
        printf("Sequential version: %lf seconds\n", cpu_time);

        double gpu_time = measure_time_func_gpu(matrixVectorMul_gpu, M, V, R_gpu, M_device, V_device, R_device, size_M, size_V, dimension);
        printf("GPU version: %lf seconds\n", gpu_time);

        if (!check_condition(R == R_gpu, "CPU and GPU results do not match"))
            continue;

        double speedup = cpu_time / gpu_time;
        printf("Speedup: %lf\n", speedup);
    }

    return 0;
}
