#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#define assertm(exp, msg) assert(((void)msg, exp))

int num_blocks, num_threads;

__global__ void matrixVectorMulKernel(int *M_d, int *V_d, int *R_d, size_t m);
void matrixVectorMul_cpu(int *M, int *V, int *R, size_t m);
void matrixVectorMul_gpu(int *M, int *V, int *R, size_t m);

void matrixVectorMul_cpu(int *M, int *V, int *R, size_t m)
{
    for (int i = 0; i < m; i++)
    {
        R[i] = 0;
        int sum_row = 0;
        for (int j = 0; j < m; j++)
        {
            sum_row += M[i * m + j] * V[j];
        }
        R[i] = sum_row;
    }
}

__global__ void matrixVectorMulKernel(int *M_d, int *V_d, int *R_d, size_t m)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = threadId; i < m; i += stride)
    {
        int sum_row = 0;
        for (int j = 0; j < m; j++)
        {
            sum_row += M_d[i * m + j] * V_d[j];
        }
        R_d[i] = sum_row;
    }
}

void matrixVectorMul_gpu(int *M, int *V, int *R, size_t m)
{
    matrixVectorMulKernel<<<num_blocks, num_threads>>>(M, V, R, m);
    hipDeviceSynchronize();
}

void fill_random(int *M, int *V, size_t m)
{
    for (int i = 0; i < m; i++)
    {
        V[i] = rand() % INT_MAX;
        for (int j = 0; j < m; j++)
        {
            M[i * m + j] = rand() % INT_MAX;
        }
    }
}

bool check_result(int *R_cpu, int *R_gpu, size_t m)
{
    for (int i = 0; i < m; i++)
    {
        if (R_cpu[i] != R_gpu[i])
        {
            printf("Error: CPU and GPU results do not match\n");
            printf("R_cpu[%d] = %d, R_gpu[%d] = %d\n", i, R_cpu[i], i, R_gpu[i]);
            return false;
        }
    }
    return true;
}

double measure_time_func_cpu(void (*func)(int *, int *, int *, size_t), int *M, int *V, int *R, size_t m)
{
    clock_t start, end;
    start = clock();
    func(M, V, R, m);
    end = clock();
    return ((double)(end - start)) / CLOCKS_PER_SEC;
}

double measure_time_func_gpu(void (*func_gpu)(int *, int *, int *, size_t), int *M, int *V, int *R, int *M_device, int *V_device, int *R_device, size_t size_M, size_t size_V, size_t m)
{
    clock_t start, end;
    hipError_t err;

    // Start timing
    start = clock();

    // Copy data to device
    err = hipMemcpy(M_device, M, size_M, hipMemcpyHostToDevice);
    assertm(err == hipSuccess, "Failed to copy matrix M from host to device");

    err = hipMemcpy(V_device, V, size_V, hipMemcpyHostToDevice);
    assertm(err == hipSuccess, "Failed to copy vector V from host to device");

    // Execute the kernel function
    func_gpu(M_device, V_device, R_device, m);
    hipDeviceSynchronize();

    // Copy result back to host
    err = hipMemcpy(R, R_device, size_V, hipMemcpyDeviceToHost);
    assertm(err == hipSuccess, "Failed to copy vector R from device to host");

    // End timing
    end = clock();

    return ((double)(end - start)) / CLOCKS_PER_SEC;
}

bool check_condition(bool cond, const char *msg)
{
    if (!cond)
        fprintf(stderr, "Error: %s\n", msg);

    return cond;
}

class IntArray
{
public:
    IntArray(size_t size) : data(nullptr), size(size)
    {
        data = (int *)malloc(size);
        if (!data)
            fprintf(stderr, "Error: Failed to allocate memory\n");
    }
    ~IntArray()
    {
        if (data)
            free(data);
    }

    int *data;
    size_t size;

    operator int *() { return data; }

    explicit operator bool() { return data != nullptr && size > 0; }
};

class IntArrayDevice
{
public:
    IntArrayDevice(size_t size) : data(nullptr), size(size)
    {
        hipError_t err = hipMalloc(&data, size);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Error: Failed to allocate CUDA memory\n");
            data = nullptr;
        }
    }
    ~IntArrayDevice()
    {
        if (data)
            hipFree(data);
    }

    int *data;
    size_t size;

    operator int *() { return data; }

    explicit operator bool() { return data != nullptr && size > 0; }
};

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        printf("Usage: %s <num_blocks> <num_threads>\n", argv[0]);
        return 1;
    }
    else
    {
        num_blocks = atoi(argv[1]);
        num_threads = atoi(argv[2]);
    }
    size_t test_dimensions[10] = {256, 512, 1024, 4096, 8192, 16384, 32768, 65536, 131072, 262144};

    for (size_t dimension : test_dimensions)
    {
        printf("Dimension: %lu\n", dimension);

        size_t size_M = dimension * dimension * sizeof(int);
        IntArray M(size_M);
        if (!check_condition(M, "Memory allocation failed for M"))
            continue;

        size_t size_V = dimension * sizeof(int);
        IntArray V(size_V);
        if (!check_condition(V, "Memory allocation failed for V"))
            continue;

        srand(time(0));
        fill_random(M, V, dimension);

        IntArray R(size_V);
        if (!check_condition(R, "Memory allocation failed for R"))
            continue;

        IntArrayDevice M_device(size_M);
        if (!check_condition(M_device, "Memory allocation failed for M_device"))
            continue;

        IntArrayDevice V_device(size_V);
        if (!check_condition(V_device, "Memory allocation failed for V_device"))
            continue;

        IntArrayDevice R_device(size_V);
        if (!check_condition(R_device, "Memory allocation failed for R_device"))
            continue;

        IntArray R_gpu(size_V);
        if (!check_condition(R_gpu, "Memory allocation failed for R_gpu"))
            continue;

        double cpu_time = measure_time_func_cpu(matrixVectorMul_cpu, M.data, V.data, R.data, dimension);
        printf("Sequential version: %f seconds\n", cpu_time);

        double gpu_time = measure_time_func_gpu(matrixVectorMul_gpu, M.data, V.data, R_gpu.data, M_device.data, V_device.data, R_device.data, size_M, size_V, dimension);
        printf("GPU version: %f seconds\n", gpu_time);

        hipError_t err = hipMemcpy(R_gpu, R_device, size_V, hipMemcpyDeviceToHost);
        if (!check_condition(err == hipSuccess, "Failed to copy vector R from device to host"))
            continue;

        if (!check_condition(check_result(R.data, R_gpu.data, dimension), "CPU and GPU results do not match"))
            continue;

        float speedup = cpu_time / gpu_time;
        printf("Speedup: %f\n", speedup);

        if (M_device)
            hipFree(M_device);
        if (V_device)
            hipFree(V_device);
        if (R_device)
            hipFree(R_device);
    }

    return 0;
}
